/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/copying.hpp>
#include <cudf/detail/search.hpp>
#include <cudf/detail/sorting.hpp>

#include <numeric>

#include <thrust/transform_scan.h>

namespace cudf {
namespace detail {

std::unique_ptr<column> index_of(
  table_view const& input,
  table_view const& keys,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_default_resource(),
  hipStream_t stream                 = 0)
{
  rmm::device_vector<size_type> idx(input.num_rows());
  thrust::sequence(rmm::exec_policy(stream)->on(stream), idx.begin(), idx.end(), 0);

  column_view idx_view(
    data_type(type_to_id<size_type>()), idx.size(), idx.data().get(), nullptr, 0);
  table_view key_table({table_view({idx_view}), input});

  auto sorted_key_table = sort_by_key(key_table, input);

  std::vector<size_type> column_selection(input.num_columns());
  std::iota(column_selection.begin(), column_selection.end(), 1);

  auto index_column = sorted_key_table->view().column(0);
  auto search_table = sorted_key_table->view().select(column_selection);

  auto lb = lower_bound(search_table,
                        keys,
                        std::vector<order>(search_table.num_columns(), order::ASCENDING),
                        std::vector<null_order>(search_table.num_columns(), null_order::AFTER),
                        mr,
                        stream);

  auto ub = upper_bound(search_table,
                        keys,
                        std::vector<order>(search_table.num_columns(), order::ASCENDING),
                        std::vector<null_order>(search_table.num_columns(), null_order::AFTER),
                        mr,
                        stream);

  // calculate size of the result
  rmm::device_vector<size_type> offsets(lb->size() + 1, 0);
  thrust::transform_inclusive_scan(
    thrust::device,
    thrust::make_zip_iterator(
      thrust::make_tuple(lb->view().begin<size_type>(), ub->view().begin<size_type>())),
    thrust::make_zip_iterator(
      thrust::make_tuple(lb->view().end<size_type>(), ub->view().end<size_type>())),
    offsets.begin() + 1,
    [] __device__(thrust::tuple<size_type, size_type> elem) -> size_type {
      return thrust::get<1>(elem) - thrust::get<0>(elem);
    },
    thrust::plus<size_type>());

  auto result_size = (size_type)offsets[lb->size()];

  auto result = cudf::make_numeric_column(
    data_type(type_to_id<size_type>()), result_size, mask_state::UNALLOCATED, stream, mr);

  auto d_result = result->mutable_view().begin<size_type>();
  auto d_idx    = index_column.begin<size_type>();

  thrust::for_each_n(
    rmm::exec_policy()->on(stream),
    thrust::make_zip_iterator(thrust::make_tuple(
      lb->view().begin<size_type>(), ub->view().begin<size_type>(), offsets.begin())),
    result_size,
    [d_idx, d_result] __device__(thrust::tuple<size_type, size_type, size_type> index_tup) {
      thrust::for_each(thrust::seq,
                       thrust::make_counting_iterator(thrust::get<0>(index_tup)),
                       thrust::make_counting_iterator(thrust::get<1>(index_tup)),
                       [d_idx, d_result, index_tup] __device__(size_type i) {
                         *(d_result + thrust::get<2>(index_tup)) = *(d_idx + i);
                       });
    });
  return result;
}

}  // namespace detail

std::unique_ptr<column> index_of(table_view const& input,
                                 table_view const& keys,
                                 rmm::mr::device_memory_resource* mr)
{
  return detail::index_of(input, keys, mr, 0);
}

}  // namespace cudf
